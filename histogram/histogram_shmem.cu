// This program shows off a shared memory implementation of a histogram
// kernel in CUDA
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <fstream>

// Number of bins for our plot
#define BINS 7
#define DIV ((26 + BINS - 1) / BINS)

using namespace std;

// GPU kernel for computing a histogram
// Takes:
//  a: Problem array in global memory
//  result: result array
//  N: Size of the array
__global__ void histogram(char *a, int *result, int N){
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Allocate a local histogram for each TB
    __shared__ int s_result[BINS];

    // Initalize the shared memory to 0
    if(threadIdx.x < BINS){
        s_result[threadIdx.x] = 0;
    }

    // Wait for shared memory writes to complete
    __syncthreads();

    // Calculate the bin positions locally
    int alpha_position;
    for(int i = tid; i < N; i += (gridDim.x * blockDim.x)){
        // Calculate the position in the alphabet
        alpha_position = a[i] - 'a';
        atomicAdd(&s_result[(alpha_position / DIV)], 1);
    }

    // Wait for shared memory writes to complete
    __syncthreads();

    // Combine the partial results
    if(threadIdx.x < BINS){
        atomicAdd(&result[threadIdx.x], s_result[threadIdx.x]);
    }
}

// Initializes our input array
// Takes:
//  a: array of integers
//  N: Length of the array
void init_array(char *a, int N){
    for(int i = 0; i < N; i++){
        a[i] = 'a' +  (rand() % 26);
    }
}

int main(){
    // Declare our problem size
    int N = 1 << 20;

    // Allocate memory on the host
    char *h_a = new char[N];
    size_t bytes_a = N * sizeof(char);

    // Allocate space for the binned result
    int *h_result = new int[BINS];
    size_t bytes_r = BINS * sizeof(int);

    // Initialize the array
    init_array(h_a, N);
    
    // Allocate memory on the device
    char *d_a;
    int *d_result;
    hipMalloc(&d_a, bytes_a);
    hipMalloc(&d_result, bytes_r);

    // Copy the array to the device
    hipMemcpy(d_a, h_a, bytes_a, hipMemcpyHostToDevice);
    hipMemcpy(d_result, h_result, bytes_r, hipMemcpyHostToDevice);

    // Number of threads per threadblock
    int THREADS = 512;

    // Calculate the number of threadblocks
    int BLOCKS = N / THREADS;

    // Launch the kernel
    histogram<<<BLOCKS, THREADS>>>(d_a, d_result, N);

    // Copy the result back
    hipMemcpy(h_result, d_result, bytes_r, hipMemcpyDeviceToHost);

    // Write the data out for gnuplot
    ofstream output_file;
    output_file.open("histogram.dat", ios::out | ios::trunc);

    for(int i = 0; i < BINS; i++){
        output_file << h_result[i] << " \n\n";
    }
    output_file.close();

    // Free memory
    delete [] h_a;
    delete [] h_result;
    hipFree(d_a);
    hipFree(d_result);

    return 0;
}
