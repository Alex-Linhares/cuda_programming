// This program computes the product of two matrices
// By: Nick from CoffeeBeforeArch

#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

__global__ void matrixMul(int *a, int *b, int *c, int n) {
	// Compute each thread's row
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	// Compute each thread's column
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int temp_sum = 0;
	// Boundary protection
	if ((row < n) && (col < n)) {
		// Iterate over row, and down column
		for (int k = 0; k < n; k++) {
			// Accumulate result for a single element
			temp_sum += a[row * n + k] + b[k * n + col];
		}
	}
	// Assign result
	c[row * n + col] = temp_sum;
}

// Initialization function for matrices
void init_matrices(int *a, int *b, int n) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			a[i * n + j] = rand() % 100;
			b[i * n + j] = rand() % 100;
		}
	}
}

// Check result
void verify_result() {

}

int main() {
	// Matrix size of 1024 x 1024;
	int n = 1 << 10;

	// Size (in bytes) of matrix
	size_t bytes = n * n * sizeof(int);

	// Host pointers
	int *h_a, *h_b, *h_c;

	// Allocate host memory
	h_a = (int*)malloc(bytes);
	h_b = (int*)malloc(bytes);
	h_c = (int*)malloc(bytes);

	// Device pointers
	int *d_a, *d_b, *d_c;

	// Allocated device memory
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	// Initialize matrices
	init_matrices(h_a, h_b, n);

	// Copy data to the device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);

	// Threads per block
	int BLOCK_SIZE = 128;

	// Blocks in each dimension
	int GRID_SIZE = (int)ceil(n / 128);

	// Use dim3 objects
	dim3 grid(GRID_SIZE, GRID_SIZE);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	// Launch kernel
	matrixMul<<<grid, threads>>>(d_a, d_b, d_c, n);

	// Copy back to the host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	// Check result
	verify_result();

	return 0;
}