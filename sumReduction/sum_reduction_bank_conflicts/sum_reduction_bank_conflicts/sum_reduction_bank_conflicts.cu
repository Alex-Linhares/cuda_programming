#include "hip/hip_runtime.h"
// This program performs sum reduction with an optimization
// removing warp divergence
// By: Nick from CoffeeBeforeArch

#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>

#define SIZE 256
#define SHMEM_SIZE 256 * 4

__global__ void sum_reduction(int *v, int *v_r) {
	// Allocate shared memory
	__shared__ int partial_sum[SHMEM_SIZE];

	// Calculate thread ID
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Load elements into shared memory
	partial_sum[threadIdx.x] = v[tid];
	__syncthreads();

	// 
	for (int s = 1; s < blockDim.x; s *= 2) {
		// Change the indexing to be sequential threads
		int index = 2 * s * threadIdx.x;

		// Each thread does work unless the index goes off the block
		if (index < blockDim.x) {
			partial_sum[index] += partial_sum[index + s];
		}
		__syncthreads();
	}

	// Let the thread 0 for this block write it's result to main memory
	// Result is inexed by this block
	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = partial_sum[0];
	}
}

void initialize_vector(int *v, int n) {
	for (int i = 0; i < n; i++) {
		v[i] = 1;//rand() % 10;
	}
}

int main() {
	// Vector size
	int n = 1 << 16;
	size_t bytes = n * sizeof(int);

	// Original vector and result vector
	int *h_v, *h_v_r;
	int *d_v, *d_v_r;

	// Allocate memory
	h_v = (int*)malloc(bytes);
	h_v_r = (int*)malloc(bytes);
	hipMalloc(&d_v, bytes);
	hipMalloc(&d_v_r, bytes);

	// Initialize vector
	initialize_vector(h_v, n);

	// Copy to device
	hipMemcpy(d_v, h_v, bytes, hipMemcpyHostToDevice);

	// TB Size
	int TB_SIZE = SIZE;

	// Grid Size (No padding)
	int GRID_SIZE = n / TB_SIZE;

	// Call kernel
	sum_reduction << <GRID_SIZE, TB_SIZE >> > (d_v, d_v_r);

	sum_reduction << <1, TB_SIZE >> > (d_v_r, d_v_r);

	// Copy to host;
	hipMemcpy(h_v_r, d_v_r, bytes, hipMemcpyDeviceToHost);

	// Print the result
	//printf("Accumulated result is %d \n", h_v_r[0]);
	//scanf("Press enter to continue: ");
	assert(h_v_r[0] == 65536);

	printf("COMPLETED SUCCESSFULLY\n");

	return 0;
}
