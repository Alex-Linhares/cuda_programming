// This program uses cuRAND and cuBLAS to perform matrix multiplication
// By: Nick from CoffeeBeforeArch

#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <time.h>
#include <assert.h>

// Verify the result (Note, cuBLAS works in column-major format)
// Assumes contiguous memory is down columns, not across rows
void verify_result(float *a, float *b, float *c, int n){
    float temp = 0.0f;
    float epsilon = 0.00001f;
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            temp = 0;
            for(int k = 0; k < n; k++){
                temp += a[k * n + i] * b[j * n + k];
            }
            assert(abs(c[j * n + i] - temp) < epsilon);
        }        
    }
}

int main(){
    // Problem size
    int n = 1 << 1024;
    size_t bytes = n * n * sizeof(float);

    // Declare pointers
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    
    // Allocate memory
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // RNG handle
    hiprandGenerator_t prng;

    // Create the Generator
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    // Initialize the matrices on the GPU
    hiprandGenerateUniform(prng, d_a, n * n);
    hiprandGenerateUniform(prng, d_b, n * n);

    // Scaling factors
    float alpha = 1;
    float beta = 0;

    // cuBLAS handle
    hipblasHandle_t handle;
    
    // Create the handle
    hipblasCreate(&handle);

    // Call SGEMM (alpha * a) * b + (beta * c)
    // Matrix definitions: op1(a) m*k, op1(b) k*n, c m*n
    // Function signature: handle, op1, op2, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n);

    // Copy back the result, and original matrices
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Verify the result
    verify_result(h_a, h_b, h_c, n);

    return 0;
}
