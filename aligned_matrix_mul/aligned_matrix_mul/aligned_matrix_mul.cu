#include "hip/hip_runtime.h"
// This program is an optimized version of Matrix Multiplication
// By: Nick from CoffeeBeforeArch

#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>

#include "kernels.cuh"

void transpose_matrix(int *a, int *a_t, int n) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			a_t[i*n + j] = a[j*n + i];
		}
	}
}

void check_answer(int *a, int *b, int *c, int n) {
	int *verify_c;
	verify_c = (int*)malloc(n * n * sizeof(int));
	int temp_val;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			temp_val = 0;
			for (int k = 0; k < n; k++) {
				temp_val += a[i * n + k] * b[k * n + j];
			}
			verify_c[i * n + j] = temp_val;
		}
	}

	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			assert(c[i * n + j] == verify_c[i * n + j]);
		}
	}
}

void init_matrix(int *a, int n) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			a[i * n + j] = rand() % 10;
		}
	}
}

int main() {
	// Problem size = 1024 x 1024 matrix
	int n = 1 << 10;

	// Matrix size (in bytes)
	size_t bytes = n * n * sizeof(int);

	// Host pointer to transposeed matrix
	int *h_a_t;

	// Host matrix pointers
	int *h_a, *h_b, *h_c;

	// Device matrix pointers
	int *d_a, *d_b, *d_c;

	// Device pointer to transposeed matrix
	int *d_a_t;

	// Allocate host memory
	h_a_t = (int*)malloc(bytes);
	h_a = (int*)malloc(bytes);
	h_b = (int*)malloc(bytes);
	h_c = (int*)malloc(bytes);

	// Allocate device memory
	hipMalloc(&d_a_t, bytes);
	//hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);


	// Initialize matrices
	init_matrix(h_a, n);
	init_matrix(h_b, n);

	// Transpose matrix a
	transpose_matrix(h_a, h_a_t, n);

	// Copy matrices to the device
	hipMemcpy(d_a_t, h_a_t, bytes, hipMemcpyHostToDevice);
	//hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	// Threads per block (in both x and y dimensions)
	int BLOCK_SIZE = 16;

	// Blocks in each dimension
	int GRID_SIZE = (int)ceil(n / BLOCK_SIZE);

	// Use dim3 objects for 2-D grids and threadblocks
	dim3 grid(GRID_SIZE, GRID_SIZE);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	// Launch kernel
	matrixMulAligned<<<grid, threads >>> (d_a_t, d_b, d_c, n);

	// Copy result back from device
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	// Verify the result
	check_answer(h_a, h_b, h_c, n);

	// Launch kernel
	//tiledMatrixMul <<<grid, threads >>> (d_a, d_b, d_c, n, BLOCK_SIZE);

	// Copy result back from device
	//hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	// Verify the result
	//check_answer(h_a, h_b, h_c, n);

	// Free host memory
	free(h_a);
	free(h_b);
	free(h_c);

	// Free device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	printf("COMPLETED SUCCESSFULLY\n");

	return 0;
}
